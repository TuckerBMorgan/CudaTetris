
#include "hip/hip_runtime.h"

/*
#include <lua.h>
#include <lauxlib.h>
#include <lualib.h>
*/
#include <stdio.h>
#include <stdint.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

//__global__ lua_State* luaL_newstate();


__global__ void addKernel(int *c, const int *a, const int *b)
{
	//lua_State* L = luaL_newstate();
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

void fill_frame(uint8_t* frame) {
}


int try_stuff(int* check)
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 11, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };
//	lua_State *L = 
	

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
	printf("hllogddffge world\n");

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,   %d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

	check[0] = c[0];
	check[1] = c[1];
	check[2] = c[2];
	printf("hlloe world\n");

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

__global__ void color_in_array(uint8_t* resultFrame, size_t screen_height) {
	int x = threadIdx.x;
	int y = threadIdx.y;

	resultFrame[x + y * screen_height] = x % 255;
}

hipError_t render_screen(uint8_t* resultFrame, size_t width, size_t height) {
	hipError_t result;
	uint8_t* screenBuffer = 0;
	
	size_t value = sizeof(uint8_t) * width * height;
	result = hipMalloc((void**)&screenBuffer, value);
	if (result != hipSuccess) {
		fprintf(stderr, "Error with the backbuffer malloc");
		hipFree(screenBuffer);//attempt to free it atleast 
		return result;
	}

	color_in_array <<<1, width * height>>> (screenBuffer, height);
	result = hipGetLastError();

	if (result != hipSuccess) {
		fprintf(stderr, "Error with the backbuffer malloc");
		hipFree(screenBuffer);//attempt to free it atleast 
		return result;
	}
	
	result = hipMemcpy(resultFrame, screenBuffer, width * height * sizeof(uint8_t), hipMemcpyDeviceToHost);

	hipFree(screenBuffer);
	return result;
	
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
